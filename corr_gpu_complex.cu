#include "hip/hip_runtime.h"
/* GPU-based correlation code for a small (<=64) input
 * sampling system to run on a workstation class computer. There is an
 * upper limit to the number of supported channels due to the max number
 * of threads that can be on the GPU. This is currently 512.
 *
 * Author: Randall Wayth. Feb, 2009.
 *
 * to compile with CUDA: nvcc -O -o corr_gpu_complex corr_gpu_complex.cu -lcufft
*/

#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <string.h>
#include <sys/time.h>
#include <math.h>
#include <ctype.h>
//#include "/usr/include/complex.h"  // nvcc stuffs up for some reason without the full path name
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <hipfft/hipfft.h>
#include <fcntl.h>

//#include "/home/rwayth/Desktop/gxc_kernels.h"

#define MAX_THREADS 128
#define MAX_INPUTS 256
#define MAX_CORR_PROD (MAX_INPUTS*(MAX_INPUTS+1)/2)
#define MAX_CHAN 128
//#define GRPSIZ 2    // this cannot be changed from 2 without rewriting the 1xG CMAC function.
#define INTMULT(a,b) __mul24((a),(b))
//#define INTMULT(a,b) (a)*(b)

#define TILE_SIZE_UNPACK 16 // note: if you change this, you must change LOG2_TILESIZE too!
#define LOG2_TILESIZE 4
#define UNPACK_BLOCK_ROWS (256/TILE_SIZE_UNPACK)

#define BITS_PER_NUM 5
#define TWOCOMP_MAXUINT   (1<<(BITS_PER_NUM))    // = 32 for 5 bits
#define TWOCOMP_ZEROPOINT (1<<(BITS_PER_NUM-1))  // = 16 for 5 bits
#define DECODER_MASK (TWOCOMP_MAXUINT -1)

#define BITS_PER_NUM_8BIT 4
#define TWOCOMP_MAXUINT_8BIT   (1<<(BITS_PER_NUM_8BIT))    // = 16 for 4 bits
#define TWOCOMP_ZEROPOINT_8BIT (1<<(BITS_PER_NUM_8BIT-1))  // = 8  for 4 bits
#define DECODER_MASK_8BIT (TWOCOMP_MAXUINT_8BIT -1)

enum {CMAC_1x1, CMAC_1xG_2, CMAC_1xG_4, CMAC_GxG_2, CMAC_GxG_4, CMAC_END};
enum {DATATYPE_MWA16, DATATYPE_MWA8, DATATYPE_COMPLEX_SHORT, DATATYPE_COMPLEX_FLOAT,
    DATATYPE_SIGNED_BYTE,DATATYPE_UNSIGNED_BYTE,DATATYPE_SIGNED_SHORT,DATATYPE_UNSIGNED_SHORT,
    DATATYPE_FLOAT,DATATYPE_COMPLEX_UNSIGNED_BYTE,DATATYPE_COMPLEX_SIGNED_BYTE,DATATYPE_REAL_UNSIGNED_4BIT};

/* function prototypes */
void print_usage(char * const argv[]);
void parse_cmdline(int argc, char * const argv[], const char *optstring);
int readData(int nchan,int ninp,FILE *fpin,unsigned char *inp_buf);
int openFiles(char *infilename, char *outfilename, int prod_type, FILE **fin, FILE **fout_ac, FILE **fout_cc);
int do_FFT_gpu(int nchan, int ninp, hipfftComplex *inp_buf, hipfftComplex *ft_buf);
void writeOutput(FILE *fout_ac, FILE *fout_cc,int ninp, int nchan, int naver, int prod_type,
                 hipFloatComplex *buf,float normaliser);
void do_CMAC_gpu(int method, const int nchan, const int ninp, const int batchsize,
         hipfftComplex * const ft_buf, hipfftComplex * const corr_buf);
__global__ void do_CMAC_gpu_1xG(const int nchan, const int ninp, const int batchsize,
         hipFloatComplex * const ft_buf, hipFloatComplex * const corr_buf);
__global__ void do_CMAC_gpu_1xG_4(const int nchan, const int ninp, const int batchsize,
         hipFloatComplex * const ft_buf, hipFloatComplex * const corr_buf);
__global__ void do_CMAC_gpu_1x1(const int nchan, const int ninp, const int batchsize,
            hipFloatComplex * const ft_buf, hipFloatComplex * const corr_buf);
__global__ void do_CMAC_gpu_GxG(const int nchan, const int ninp, const int batchsize,
            hipFloatComplex * const ft_buf, hipFloatComplex * const corr_buf);
__global__ void do_CMAC_gpu_GxG_4(const int nchan, const int ninp, const int batchsize,
            hipFloatComplex * const ft_buf, hipFloatComplex * const corr_buf);
__global__ void unpack_data_GPU_MWA16bit_transpose(const int nchan,const int ninp,const int batchsize, int log2_nchan,
                unsigned short *in_buf, hipFloatComplex *out_buf);
__global__ void unpack_data_GPU_MWA16bit_simple(const int nchan,const int ninp,const int batchsize, unsigned short *staging_buf,
                hipFloatComplex *inp_buf);
__global__ void unpack_data_GPU_16bit_simple(const int nchan,const int ninp,const int batchsize, char2 *staging_buf,
                hipFloatComplex *inp_buf);
__global__ void unpack_data_GPU_uint8_complex_simple(const int nchan,const int ninp,const int batchsize, char2 *staging_buf,
                hipFloatComplex *inp_buf);
__global__ void unpack_data_GPU_MWA8bit_simple(const int nchan,const int ninp,const int batchsize, unsigned char *staging_buf,
                hipFloatComplex *inp_buf_gpu);
__global__ void unpack_data_GPU_MWA8bit_transpose(const int nchan,const int ninp,const int batchsize, int log2_nchan,
                unsigned char *in_buf, hipFloatComplex *out_buf);
static float elapsed_time(struct timeval *start);
void printGPUDetails(FILE *fp);
int init_CMAC_1x1(void);
int init_CMAC_GxG(const int);

/* global vars */
int nchan=128;
int ninp=16;
int debug=1;
int naver=10000;
int fft_batchsize=25;
int wordtype=DATATYPE_COMPLEX_SIGNED_BYTE,wordsize=0,complex_data=1; // input data format type
int unpack_method=0;
int cmac_method=CMAC_GxG_4;
char *infilename=NULL,*outfilename=NULL;
int prod_type='B';  /* correlation product type: B: both, C: cross, A: auto */
//__device__ __constant__ uint2 inp_index_gpu[MAX_CORR_PROD]; // cannot fit for 256 inputs
__device__ uint2 inp_index_gpu[MAX_CORR_PROD];

int main(int argc, char * const argv[]) {
    size_t siz_inp_buf=0,siz_ft_buf=0,siz_corr_buf=0,siz_inp_gpu=0;
    int i,nmoves=0,iter=0,nav_written=0,log2_nchan;
    int filedone=0,ncorr;
    FILE *finp=NULL,*fout_ac=NULL,*fout_cc=NULL;
    char optstring[]="c:i:o:n:a:f:p:w:m:u:";
    unsigned char *staging_buf=NULL,*staging_buf_gpu=NULL;
    hipfftComplex *inp_buf_gpu=NULL;
    hipfftComplex *ft_buf=NULL;
    hipFloatComplex *corr_buf=NULL;
    hipFloatComplex *corr_buf_gpu=NULL;
    hipError_t res;
    struct timeval thetime,starttime;
    float read_time=0,fft_time=0,cmac_time=0,sync_time=0,write_time=0,move_time=0,unpack_time=0;
    dim3 gridUnpack,blockUnpack;

    
    gettimeofday(&starttime,NULL);

    /* process command line args */
    if (argc <2) print_usage(argv);
    parse_cmdline(argc,argv,optstring);
    ncorr = ninp*(ninp+1)/2;

    /* check that nchan is power of 2 */
    for(i=1;i<30;i++) {
        if(1<<i == nchan) {
            log2_nchan=i;
            break;
        }
    }
    if (log2_nchan ==0) {
        fprintf(stderr,"ERROR: number of channels must be a power of 2\n");
    }

    /* calculate the size of various buffers in bytes */
    siz_inp_buf  = ninp*nchan*wordsize*fft_batchsize;
    siz_inp_gpu  = ninp*nchan*sizeof(hipfftComplex)*fft_batchsize;
    siz_ft_buf   = ninp*nchan*sizeof(hipfftComplex)*fft_batchsize;
    siz_corr_buf = ncorr*nchan*sizeof(hipfftComplex);

    if (debug) {
        printGPUDetails(stderr);
        fprintf(stderr,"---\nNum inp:\t%d. Num corr products: %d\n",ninp,ncorr);
        fprintf(stderr,"CMAC method:\t%d\n",cmac_method);
        fprintf(stderr,"Num chan:\t%d\n",nchan);
        fprintf(stderr,"infile: \t%s\n",infilename);
        fprintf(stderr,"outfile:\t%s\n",outfilename);
        fprintf(stderr,"batchsize:\t%d\n",fft_batchsize);
        fprintf(stderr,"Input buffer size:\t%ld\n",(long)siz_inp_buf);
        //fprintf(stderr,"Size of uchar: %d\n",sizeof(unsigned char));
    }
    
    /* open input and output files */
    openFiles(infilename,outfilename,prod_type, &finp,&fout_ac,&fout_cc);

    /* allocate buffers */
    /* input buffer on host, which gets transferred to the GPU */
    res = hipHostMalloc((void **)&staging_buf,siz_inp_buf);
    if (res != 0) {
        fprintf(stderr,"failed to alloc host mem for staging_buf. Error: %s\n",hipGetErrorString(res));
        exit(1);
    }
    
    /* staging buffer, which lives on the GPU. this holds packed (byte) data */
    res = hipMalloc((void **)&staging_buf_gpu,siz_inp_buf);
    if (res != 0) {
        fprintf(stderr,"failed to alloc device mem for staging_buf_gpu\n");
        exit(1);
    }
    
    /* input buffer, which lives on the GPU. This holds unpacked (float) data */
    res = hipMalloc((void **)&inp_buf_gpu,siz_inp_gpu);
    if (res != 0) {
        fprintf(stderr,"failed to alloc device mem for inp_buf_gpu\n");
        exit(1);
    }

    /* FFT result buffer, lives on GPU only */
    res = hipMalloc((void **)&ft_buf,siz_ft_buf);
    if (res != 0) {
        fprintf(stderr,"failed to alloc device mem for ft_buf\n");
        exit(1);
    }

    /* for correlation products */
    /* results of CMAC on the GPU */
    res=hipMalloc((void **)&corr_buf_gpu,siz_corr_buf);
    if (res != 0) {
        fprintf(stderr,"failed to alloc device mem for corr_buf_gpu\n");
        exit(1);
    }

    /* space on the host for CMAC results to be transferred to */
    res=hipHostMalloc((void **)&corr_buf,siz_corr_buf);
    if (res != 0) {
        fprintf(stderr,"failed to alloc host mem for corr_buf\n");
        exit(1);
    }

    /* init to zero, since stuff is accumulated into these arrays */
    hipMemset(corr_buf_gpu,'\0',siz_corr_buf);
    hipMemset(ft_buf,'\0',siz_ft_buf);
    hipMemset(inp_buf_gpu,'\0',siz_inp_gpu);

    if ( ((nchan*ninp*fft_batchsize) % MAX_THREADS) != 0) {
        fprintf(stderr,"can't find integer mult of threads for unpack\n");
        exit(1);
    }

    /* execution configuration for the unpack/corner turn stage */
    if(unpack_method==0) {
        /* simple unpack method: each thread writes a time/channel for an input.
           Grid dimension X is over all inputs, then Y is for time */
        gridUnpack.x = ninp;
        gridUnpack.y = fft_batchsize;
        blockUnpack.x = nchan;
    }
    else {
        /* unpack using using shared memory, which makes reads and writes coalesced.
            Note that this is slower for older GPUs, as explained below */
        gridUnpack.x = ninp/TILE_SIZE_UNPACK;
        gridUnpack.y = fft_batchsize*nchan/TILE_SIZE_UNPACK;
        blockUnpack.x = TILE_SIZE_UNPACK;
        blockUnpack.y = UNPACK_BLOCK_ROWS;        
    }

    /* process file */
    while (!filedone) {
    
        /* read time chunk into buffers on host */
        gettimeofday(&thetime,NULL);
        if (readData(nchan, ninp,finp,staging_buf) !=0) {
            filedone=1;
        }
        read_time += elapsed_time(&thetime);
        
        if (!filedone) {

            /* wait for any running correlation threads to finish */
            gettimeofday(&thetime,NULL);
            hipDeviceSynchronize();
            sync_time += elapsed_time(&thetime);
            if ( (res=hipGetLastError()) != hipSuccess) {
                fprintf(stderr,"do_CMAC_gpu failed. Error: %s\n",hipGetErrorString(res));
                goto EXIT;
            }

            /* move new input to GPU */
            gettimeofday(&thetime,NULL);
            //if (debug) fprintf(stdout,"Moving batch %d to GPU\n",iter);
            res = hipMemcpy(staging_buf_gpu,staging_buf,siz_inp_buf,hipMemcpyHostToDevice);
            if (res != hipSuccess) {
                fprintf(stderr,"Error on memcpy of data from host to device. Message: %s\n",hipGetErrorString(res));
                goto EXIT;
            }
            nmoves++;
            move_time += elapsed_time(&thetime);

            /* unpack the data into float format */
            gettimeofday(&thetime,NULL);
            switch (wordtype) {
                case DATATYPE_COMPLEX_SIGNED_BYTE:
                    unpack_data_GPU_16bit_simple<<<gridUnpack,blockUnpack>>>(nchan,ninp,fft_batchsize, (char2 *)staging_buf_gpu, inp_buf_gpu);
                    break;
                case DATATYPE_COMPLEX_UNSIGNED_BYTE:
                    unpack_data_GPU_uint8_complex_simple<<<gridUnpack,blockUnpack>>>(nchan,ninp,fft_batchsize, (char2 *)staging_buf_gpu, inp_buf_gpu);
                    break;
                case DATATYPE_MWA16:
                    if (unpack_method==1) {
                        unpack_data_GPU_MWA16bit_transpose<<<gridUnpack,blockUnpack>>>(nchan,ninp,fft_batchsize,log2_nchan,
                                         (unsigned short *)staging_buf_gpu, inp_buf_gpu);
                    }
                    else {
                        unpack_data_GPU_MWA16bit_simple<<<gridUnpack,blockUnpack>>>(nchan,ninp,fft_batchsize, (unsigned short *)staging_buf_gpu, inp_buf_gpu);
                    }
                    break;
                case DATATYPE_MWA8:
                    if (unpack_method==1) {
                        unpack_data_GPU_MWA8bit_transpose<<<gridUnpack,blockUnpack>>>(nchan,ninp,fft_batchsize,log2_nchan,
                                         (unsigned char *)staging_buf_gpu, inp_buf_gpu);
                    }
                    else {
                        unpack_data_GPU_MWA8bit_simple<<<gridUnpack,blockUnpack>>>(nchan,ninp,fft_batchsize, (unsigned char *)staging_buf_gpu, inp_buf_gpu);
                    }
                    break;
                default:
                    fprintf(stderr,"unknown data word type %d\n",wordtype);
                    exit(1);
            }
            hipDeviceSynchronize(); /* must wait before starting FFT */
            unpack_time += elapsed_time(&thetime);            
            if ( (res=hipGetLastError()) != hipSuccess) {
                fprintf(stderr,"unpack_data_GPU failed. Error: %s\n",hipGetErrorString(res));
                goto EXIT;
            }

            /* do the FFT */
            gettimeofday(&thetime,NULL);
            if (do_FFT_gpu(nchan,ninp,inp_buf_gpu,ft_buf) != HIPFFT_SUCCESS) goto EXIT;
            hipDeviceSynchronize();
            fft_time += elapsed_time(&thetime);

            /* do the CMAC. don't sync after this call since the next batch of data can be read in parallel. */
            gettimeofday(&thetime,NULL);
            do_CMAC_gpu(cmac_method,nchan,ninp,fft_batchsize, ft_buf, corr_buf_gpu);
            cmac_time += elapsed_time(&thetime);
            iter++;
        }
        
        /* write out if it is time to */
//        if ( (filedone && iter>0) || (iter*fft_batchsize >= naver) ) {
        if ( iter*fft_batchsize >= naver ) {
            hipDeviceSynchronize();
            gettimeofday(&thetime,NULL);
            /* fetch the accumulated results from the GPU */
            res=hipMemcpy(corr_buf,corr_buf_gpu,siz_corr_buf,hipMemcpyDeviceToHost);
            if (res != hipSuccess) {
                fprintf(stderr,"Error on memcpy of results from device to host. Message: %s\n",hipGetErrorString(res));
                goto EXIT;
            }
 
            /* reset to zero, since stuff is accumulated into these arrays */
            res=hipMemset(corr_buf_gpu,'\0',siz_corr_buf);
            if (res != hipSuccess) {
                fprintf(stderr,"Error on memset on host. Message: %s\n",hipGetErrorString(res));
                goto EXIT;
            }

            gettimeofday(&thetime,NULL);
            writeOutput(fout_ac,fout_cc,ninp,nchan,iter,prod_type,corr_buf,1.0/(nchan*iter*fft_batchsize));
            if(debug) fprintf(stderr,"writing average of %d chunks\n",iter*fft_batchsize);
            iter=0;
            nav_written++;
            write_time += elapsed_time(&thetime);
        }
    }
    
    if (debug) {
        fprintf(stderr,"wrote %d averages. unused chunks: %d\n",nav_written,iter*fft_batchsize);
        fprintf(stderr,"Time reading:\t%g ms (done in parallel with CMAC)\n",read_time);
        fprintf(stderr,"Time moving:\t%g ms. N moves: %d, BW: %g GB/s\n",
                move_time,nmoves,(float)nmoves*(float)siz_inp_buf/(move_time*1e-3)*1e-9);
        fprintf(stderr,"Time unpacking:\t%g ms\n",unpack_time);
        fprintf(stderr,"Time FFTing:\t%g ms\n",fft_time);
        fprintf(stderr,"Time CMACing:\t%g ms (including read-time)\n",cmac_time+sync_time+read_time);
        fprintf(stderr,"Time writing:\t%g ms\n",write_time);
        fprintf(stderr,"Total time:\t%g ms\n",elapsed_time(&starttime));
    }

EXIT:
    /* clean up */
    fclose(finp);
    if(fout_ac !=NULL) fclose(fout_ac);
    if(fout_cc !=NULL) fclose(fout_cc);
    if (staging_buf_gpu != NULL) hipFree(staging_buf_gpu);
    if (staging_buf != NULL) hipHostFree(staging_buf);
    if (ft_buf != NULL) hipFree(ft_buf);
    if (inp_buf_gpu != NULL) hipFree(inp_buf_gpu);
    if (corr_buf != NULL) hipHostFree(corr_buf);
    if (corr_buf_gpu != NULL) hipFree(corr_buf_gpu);
    return 0;
}


/* unpack the byte data into float complex format on the GPU.
   The input is a sequence of n-byte words, one for each input for each time instant. input i, time t
   i.e. i0t0,i1t0,i2t0,i3t0,i0t1,i1t1,i2t1,i3t1,i0t2, etc.
   We need to re-order these into
   sequences for inputs in the FFT. i.e.
   i0t0,i0t1,...i0,t(nchan-1),i1,t0,i1,t1,...i1,t(nchan-1)
   least significant bits are reals, then imags
*/
/* unpack data - faster method with coalesced reads and writes
   this is for MWA 16-bit format (5+5 real, imag)
   This is based on the "transposeNew" project that comes with the CUDA SDK.
   read a block of TILE_SIZE_UNPACK*TILE_SIZE_UNPACK values into shared memory
   where threads read in the "Y" direction, then write the block into the result
   array where threads write in the "X" direction.
   
   Note that for older NVIDIA GPUs (version 1.0 and 1.1), this transpose is actually slower
   than the simple method. This is becuase (as far as I can tell), the reads are not actually
   coalesced because they are shorts or chars, not a 4-byte data type.
*/
__global__ void unpack_data_GPU_MWA16bit_transpose(const int nchan,const int ninp,const int batchsize, int log2_nchan,
                unsigned short *in_buf, hipFloatComplex *out_buf) {
    int i, batch=0, sample, real, imag, x_ind, y_ind, index_in, index_out,blk_y;
    int nchan_on_tilesize;
    __shared__ hipFloatComplex tile[TILE_SIZE_UNPACK][TILE_SIZE_UNPACK+1];

    //batch = blockIdx.y*TILE_SIZE_UNPACK/nchan;
    batch = (blockIdx.y*TILE_SIZE_UNPACK)>>log2_nchan;
    nchan_on_tilesize = nchan>>LOG2_TILESIZE;
    //blk_y = blockIdx.y%(nchan/TILE_SIZE_UNPACK); // use faster division for powers of 2 below.
    blk_y = blockIdx.y&(nchan_on_tilesize-1);
    
    // find array index for this thread in input array.    
    x_ind = blockIdx.x*TILE_SIZE_UNPACK + threadIdx.x;
    y_ind = blk_y*TILE_SIZE_UNPACK + threadIdx.y;
    index_in = batch*nchan*ninp + ninp*y_ind + x_ind;

    // likewise for output array
    x_ind = blk_y*TILE_SIZE_UNPACK + threadIdx.x;
    y_ind = blockIdx.x*TILE_SIZE_UNPACK + threadIdx.y;
    index_out = batch*nchan*ninp + nchan*y_ind + x_ind;

    for (i=0; i<TILE_SIZE_UNPACK ; i+=UNPACK_BLOCK_ROWS) {
        sample = in_buf[index_in + i*ninp];
        real = sample&DECODER_MASK;
        imag = (sample>>BITS_PER_NUM)&DECODER_MASK;
        if(imag >= TWOCOMP_ZEROPOINT) {
            imag -= TWOCOMP_MAXUINT;
        }
        if(real >= TWOCOMP_ZEROPOINT) {
            real -= TWOCOMP_MAXUINT;
        }
        tile[threadIdx.y+i][threadIdx.x] = make_hipFloatComplex(real,imag);
    }
    __syncthreads();
#ifdef __DEVICE_EMULATION__
    //printf("blk: %d, thr: %d, outind: %d, batind: %d, ibi: %d, inpind: %d, timind: %d, sampind: %d, sample: %d, real: %d, imag: %d\n", blockIdx.x,threadIdx.x,output_index,batch_index,intra_batch_index,input_index,time_index,sample_index,sample,real,imag); 
#endif
    for (i=0; i<TILE_SIZE_UNPACK ; i+=UNPACK_BLOCK_ROWS) {
        out_buf[index_out + i*nchan] = tile[threadIdx.x][threadIdx.y + i];
    }
}

/* unpack data - faster method with coalesced reads and writes
   this is for MWA 8-bit format (4+4 real, imag)
   This is based on the "transposeNew" project that comes with the CUDA SDK.
   read a block of TILE_SIZE_UNPACK*TILE_SIZE_UNPACK values into shared memory
   where threads read in the "Y" direction, then write the block into the result
   array where threads write in the "X" direction.
*/
__global__ void unpack_data_GPU_MWA8bit_transpose(const int nchan,const int ninp,const int batchsize, int log2_nchan,
                unsigned char *in_buf, hipFloatComplex *out_buf) {
    int i, batch=0, sample, real, imag, x_ind, y_ind, index_in, index_out,blk_y;
    int nchan_on_tilesize;
    __shared__ hipFloatComplex tile[TILE_SIZE_UNPACK][TILE_SIZE_UNPACK+1];

    //batch = blockIdx.y*TILE_SIZE_UNPACK/nchan;
    batch = (blockIdx.y*TILE_SIZE_UNPACK)>>log2_nchan;
    nchan_on_tilesize = nchan>>LOG2_TILESIZE;
    //blk_y = blockIdx.y%(nchan/TILE_SIZE_UNPACK); // use faster division for powers of 2 below.
    blk_y = blockIdx.y&(nchan_on_tilesize-1);
    
    // find array index for this thread in input array.    
    x_ind = blockIdx.x*TILE_SIZE_UNPACK + threadIdx.x;
    y_ind = blk_y*TILE_SIZE_UNPACK + threadIdx.y;
    index_in = batch*nchan*ninp + ninp*y_ind + x_ind;

    // likewise for output array
    x_ind = blk_y*TILE_SIZE_UNPACK + threadIdx.x;
    y_ind = blockIdx.x*TILE_SIZE_UNPACK + threadIdx.y;
    index_out = batch*nchan*ninp + nchan*y_ind + x_ind;

    for (i=0; i<TILE_SIZE_UNPACK ; i+=UNPACK_BLOCK_ROWS) {
        sample = in_buf[index_in + i*ninp];
        real = sample&DECODER_MASK_8BIT;
        imag = (sample>>BITS_PER_NUM_8BIT)&DECODER_MASK_8BIT;
        if(imag >= TWOCOMP_ZEROPOINT_8BIT) {
            imag -= TWOCOMP_MAXUINT_8BIT;
        }
        if(real >= TWOCOMP_ZEROPOINT_8BIT) {
            real -= TWOCOMP_MAXUINT_8BIT;
        }
        tile[threadIdx.y+i][threadIdx.x] = make_hipFloatComplex(real,imag);
    }
    __syncthreads();
#ifdef __DEVICE_EMULATION__
    //printf("blk: %d, thr: %d, outind: %d, batind: %d, ibi: %d, inpind: %d, timind: %d, sampind: %d, sample: %d, real: %d, imag: %d\n", blockIdx.x,threadIdx.x,output_index,batch_index,intra_batch_index,input_index,time_index,sample_index,sample,real,imag); 
#endif
    for (i=0; i<TILE_SIZE_UNPACK ; i+=UNPACK_BLOCK_ROWS) {
        out_buf[index_out + i*nchan] = tile[threadIdx.x][threadIdx.y + i];
    }
}


/* unpack data - simple method with non-coalesced reads, coalesced writes
   this is for MWA 16-bit format (5+5 real, imag)
*/
__global__ void unpack_data_GPU_MWA16bit_simple(const int nchan,const int ninp,const int batchsize, unsigned short *staging_buf,
                hipFloatComplex *inp_buf) {
    int out_index,inp_index;
    int sample,real,imag;
    
    inp_index = blockIdx.y*ninp*nchan + threadIdx.x*ninp + blockIdx.x;
    out_index = blockIdx.y*ninp*nchan + blockIdx.x*nchan + threadIdx.x;
    sample = staging_buf[inp_index];
    real = sample&DECODER_MASK;
    imag = (sample>>BITS_PER_NUM)&DECODER_MASK;
    if(imag >= TWOCOMP_ZEROPOINT) {
        imag -= TWOCOMP_MAXUINT;
    }
    if(real >= TWOCOMP_ZEROPOINT) {
        real -= TWOCOMP_MAXUINT;
    }
    
#ifdef __DEVICE_EMULATION__
    /*
    printf("blk: %d, thr: %d, outind: %d, batind: %d, ibi: %d, inpind: %d, timind: %d, sampind: %d, sample: %d, real: 
%d, imag: %d\n", blockIdx.x,threadIdx.x,output_index,batch_index,intra_batch_index,input_index,time_index,sample_index,s
ample,real,imag);
*/
     
#endif
    inp_buf[out_index] = make_hipFloatComplex(real,imag);
}

/* unpack data - simple method with non-coalesced reads, coalesced writes
   this is for raw 16-bit format (8+8 real, imag)
*/
__global__ void unpack_data_GPU_16bit_simple(const int nchan,const int ninp,const int batchsize, char2 *staging_buf,
                hipFloatComplex *inp_buf) {
    int out_index,inp_index;
    char2 sample;
    
    inp_index = blockIdx.y*ninp*nchan + threadIdx.x*ninp + blockIdx.x;
    out_index = blockIdx.y*ninp*nchan + blockIdx.x*nchan + threadIdx.x;
    sample = staging_buf[inp_index];
    
#ifdef __DEVICE_EMULATION__
    /*
    printf("blk: %d, thr: %d, outind: %d, batind: %d, ibi: %d, inpind: %d, timind: %d, sampind: %d, real: 
%d, imag: %d\n", blockIdx.x,threadIdx.x,output_index,batch_index,intra_batch_index,input_index,time_index,sample_index,sample.x,sample.y);
*/
     
#endif
    inp_buf[out_index] = make_hipFloatComplex(sample.x,sample.y);
}

/* unpack data - simple method with non-coalesced reads, coalesced writes
   this is for raw unsigned 8+8 real, imag
*/
__global__ void unpack_data_GPU_uint8_complex_simple(const int nchan,const int ninp,const int batchsize, char2 *staging_buf,
                hipFloatComplex *inp_buf) {
    int out_index,inp_index;
    char2 sample;
    
    inp_index = blockIdx.y*ninp*nchan + threadIdx.x*ninp + blockIdx.x;
    out_index = blockIdx.y*ninp*nchan + blockIdx.x*nchan + threadIdx.x;
    sample = staging_buf[inp_index];
    
#ifdef __DEVICE_EMULATION__
    /*
    printf("blk: %d, thr: %d, outind: %d, batind: %d, ibi: %d, inpind: %d, timind: %d, sampind: %d, real: 
%d, imag: %d\n", blockIdx.x,threadIdx.x,output_index,batch_index,intra_batch_index,input_index,time_index,sample_index,sample.x,sample.y);
*/
     
#endif
    inp_buf[out_index] = make_hipFloatComplex(sample.x-128.0,sample.y-128.0);
}




/* unpack data - simple method with non-coalesced reads, coalesced writes
   this is for MWA 8-bit format (4+4 real, imag)
*/
__global__ void unpack_data_GPU_MWA8bit_simple(const int nchan,const int ninp,const int batchsize, unsigned char *staging_buf,
                hipFloatComplex *inp_buf) {
    int out_index,inp_index;
    int sample,real,imag;
    
    inp_index = blockIdx.y*ninp*nchan + threadIdx.x*ninp + blockIdx.x;
    out_index = blockIdx.y*ninp*nchan + blockIdx.x*nchan + threadIdx.x;
    sample = staging_buf[inp_index];
    real = sample&DECODER_MASK_8BIT;
    imag = (sample>>BITS_PER_NUM_8BIT)&DECODER_MASK_8BIT;
    if(imag >= TWOCOMP_ZEROPOINT_8BIT) {
        imag -= TWOCOMP_MAXUINT_8BIT;
    }
    if(real >= TWOCOMP_ZEROPOINT_8BIT) {
        real -= TWOCOMP_MAXUINT_8BIT;
    }

#ifdef __DEVICE_EMULATION__
    printf("8bit blk: %d, thr: %d, outind: %d, batind: %d, ibi: %d, inpind: %d, timind: %d, sampind: %d, sample: %d, real: %d, imag: %d\n", blockIdx.x,threadIdx.x,output_index,batch_index,intra_batch_index,input_index,time_index,sample_index,sample,(int)real,(int)imag);
#endif
    //inp_buf[out_index] = make_hipFloatComplex(real,imag);
    inp_buf[out_index] = make_hipFloatComplex((real+0.5),(imag+0.5));
}


/* wrapper function for GPU CMAC methods */
void do_CMAC_gpu(int method, const int nchan, const int ninp, const int batchsize,
            hipfftComplex * const ft_buf, hipfftComplex * const corr_buf) {
    static int initialised=0;
    
    switch(method) {
    
        case CMAC_1xG_2: {
            /* the CMAC is executed as a 2D grid of threads. The i axis has ninp
                blocks, the j axis has ninp/2 blocks .*/
            dim3 dimBlock(nchan);
            dim3 dimGrid(ninp,ninp/2);
            do_CMAC_gpu_1xG<<<dimGrid,dimBlock>>>(nchan,ninp,fft_batchsize,
                                    (hipFloatComplex *) ft_buf,(hipFloatComplex *) corr_buf);
            break;
            }
        case CMAC_1xG_4: {
            /* the CMAC is executed as a 2D grid of threads. The i axis has ninp
                blocks, the j axis has ninp/4 blocks .*/
            dim3 dimBlock(nchan);
            dim3 dimGrid(ninp,ninp/4);
            do_CMAC_gpu_1xG_4<<<dimGrid,dimBlock>>>(nchan,ninp,fft_batchsize,
                                    (hipFloatComplex *) ft_buf,(hipFloatComplex *) corr_buf);
            break;
            }
        case CMAC_1x1: {
            if (!initialised) {
                init_CMAC_1x1();
                initialised=1;
            }
            dim3 dimBlock(nchan);
            dim3 dimGrid(ninp*(ninp+1)/2);
            do_CMAC_gpu_1x1<<<dimGrid,dimBlock>>>(nchan,ninp,fft_batchsize,
                                    (hipFloatComplex *) ft_buf,(hipFloatComplex *) corr_buf);
            
            break;
            }
        case CMAC_GxG_2: {
            if (!initialised) {
                init_CMAC_GxG(2);
                initialised=1;
            }
            dim3 dimBlock(MAX_CHAN/2,2);
            dim3 dimGrid(nchan/(MAX_CHAN/2),(ninp/2)*((ninp/2)+1)/2);
            do_CMAC_gpu_GxG<<<dimGrid,dimBlock>>>(nchan,ninp,fft_batchsize,
                                    (hipFloatComplex *) ft_buf,(hipFloatComplex *) corr_buf);
            break;
            }
        case CMAC_GxG_4: {
            if (!initialised) {
                init_CMAC_GxG(4);
                initialised=1;
            }
            dim3 dimBlock(MAX_CHAN/4,4);
            dim3 dimGrid(nchan/(MAX_CHAN/4),(ninp/4)*((ninp/4)+1)/2);
            do_CMAC_gpu_GxG_4<<<dimGrid,dimBlock>>>(nchan,ninp,fft_batchsize,
                                    (hipFloatComplex *) ft_buf,(hipFloatComplex *) corr_buf);
            break;
            }
        default: {
            fprintf(stderr,"do_CMAC_gpu: unknown CMAC method %d\n",method);
            exit(1);
            }
    }
}

 
/*  Do the CMAC on the GPU. At this stage, we have a contiguous buffer of dim [fft_batchsize][ninp][nchan]
    complex numbers in ft_buf. We need to accumulate these into correlation products in corr_buf.
    The output buffer is hipFloatComplex[ninp*(ninp+1)/2][nchan]
*/
/*
    The 1xG design uses shared memory to reduce the total number of global reads. The execution grid
    is ninp x (ninp/GRPSIZ) where GRPSIZ=2. The i index is the input index on the horizontal axis,
    the j index is the input index on the vertical axis and we want to compute the lower half of
    the correlation matrix. This means only compute products with i >= j.
    
    The kernel shares a value on the i axis between GRPSIZ (2) products on the j axis. This means that
    instead of 2*GRPSIZ reads, we do GRPSIZ+1 reads for GRPSIZ products.

    Shared memory is used to store the common value from the i index, registers are used to form the
    GRPSIZ accumulation products on the j index.
    
    The advantage of this approach over the group parallel approach is that all threads remain
    independent and no synchronisation points are required. 
*/
/* this version hard coded for GRPSIZ = 2*/
__global__ void do_CMAC_gpu_1xG(const int nchan, const int ninp, const int batchsize,
            hipFloatComplex * const ft_buf, hipFloatComplex * const corr_buf) {

    /* define inputs (a,b) and result (c) */
    __shared__ hipFloatComplex b[MAX_CHAN];
    hipFloatComplex c1,c0;
    int j = INTMULT(blockIdx.y,2);

    /* skip redundant input combinations. This is the same for all threads,
       so should be fast and inexpensive. */
    if (j > blockIdx.x) return;

    /* initialise accumulators */
    c1 = c0 = make_hipFloatComplex(0.0,0.0);

    /* loop over all the time indices (one for each FFT batch) since they all go into the same product */
    for(int batch_ind=0; batch_ind<batchsize; batch_ind++) {
        hipFloatComplex a;
        
        /* fetch the i index input, which is the same over the group */
        b[threadIdx.x] = hipConjf(ft_buf[INTMULT((INTMULT(batch_ind,ninp)+blockIdx.x),nchan) + threadIdx.x]);

        // fetch. j is always <= blockIdx.x for first product. 
        a = ft_buf[INTMULT((INTMULT(batch_ind,ninp) + j),nchan) + threadIdx.x];
        c0 = hipCaddf(c0,hipCmulf(a,b[threadIdx.x]));

        // second product. might be redundant
        a = ft_buf[INTMULT((INTMULT(batch_ind,ninp) + j+1),nchan) + threadIdx.x];
        c1 = hipCaddf(c1,hipCmulf(a,b[threadIdx.x]));
    }
    
    /* the following formula decodes the correlation index (including autocorrelations) from the input
       corr_index = blockIdx.y*(ninp+1) - blockIdx.y*(blockIdx.y+1)/2 + (blockIdx.x-blockIdx.y);
       indices x and y */
    int corr_index;

    // be careful with j*(j+1)/2. Need to do div by 2 last.
    corr_index = INTMULT(j,ninp) - INTMULT(j,(j+1))/2 + blockIdx.x;
    // accumulate and store result 
    corr_index = INTMULT(corr_index,nchan) + threadIdx.x;
    corr_buf[corr_index] = hipCaddf(corr_buf[corr_index],c0);
    
    // don't store redundant pairs
    if (++j > blockIdx.x) return;

    corr_index = INTMULT(j,ninp) - INTMULT(j,(j+1))/2 + blockIdx.x;
    corr_index = INTMULT(corr_index,nchan) + threadIdx.x;
    corr_buf[corr_index] = hipCaddf(corr_buf[corr_index],c1);
}


/* same again, but for explicit G=4 */
__global__ void do_CMAC_gpu_1xG_4(const int nchan, const int ninp, const int batchsize,
            hipFloatComplex * const ft_buf, hipFloatComplex * const corr_buf) {

    /* define inputs (a,b) and result (c) */
    __shared__ hipFloatComplex b[MAX_CHAN];
    hipFloatComplex c[4];
    int j = INTMULT(blockIdx.y,4);

    /* skip redundant input combinations. This is the same for all threads,
       so should be fast and inexpensive. */
    if (j > blockIdx.x) return;

    /* initialise accumulators */
    c[0] = c[1] = c[2] = c[3] = make_hipFloatComplex(0.0,0.0);

    /* loop over all the time indices (one for each FFT batch) since they all go into the same product */
    for(int batch_ind=0; batch_ind<batchsize; batch_ind++) {
        hipFloatComplex a;
        
        /* fetch the i index input, which is the same over the group */
        b[threadIdx.x] = hipConjf(ft_buf[INTMULT((INTMULT(batch_ind,ninp)+blockIdx.x),nchan) + threadIdx.x]);

        // fetch j 
        a = ft_buf[INTMULT((INTMULT(batch_ind,ninp) + j),nchan) + threadIdx.x];
        c[0] = hipCaddf(c[0],hipCmulf(a,b[threadIdx.x]));

        // second product. might be redundant
        a = ft_buf[INTMULT((INTMULT(batch_ind,ninp) + j+1),nchan) + threadIdx.x];
        c[1] = hipCaddf(c[1],hipCmulf(a,b[threadIdx.x]));
        
        // third product. might be redundant
        a = ft_buf[INTMULT((INTMULT(batch_ind,ninp) + j+2),nchan) + threadIdx.x];
        c[2] = hipCaddf(c[2],hipCmulf(a,b[threadIdx.x]));

        // fourth product. might be redundant
        a = ft_buf[INTMULT((INTMULT(batch_ind,ninp) + j+3),nchan) + threadIdx.x];
        c[3] = hipCaddf(c[3],hipCmulf(a,b[threadIdx.x]));
    }

    /* the following formula decodes the correlation index (including autocorrelations) from the input
       corr_index = blockIdx.y*(ninp+1) - blockIdx.y*(blockIdx.y+1)/2 + (blockIdx.x-blockIdx.y);
       indices x and y */
    int corr_index;

    // be careful with j*(j+1)/2. Need to do div by 2 last.
    corr_index = INTMULT(j,ninp) - INTMULT(j,(j+1))/2 + blockIdx.x;
    // accumulate and store result 
    corr_index = INTMULT(corr_index,nchan) + threadIdx.x;
    corr_buf[corr_index] = hipCaddf(corr_buf[corr_index],c[0]);
    
    // don't store redundant pairs
    if (++j > blockIdx.x) return;

    corr_index = INTMULT(j,ninp) - INTMULT(j,(j+1))/2 + blockIdx.x;
    corr_index = INTMULT(corr_index,nchan) + threadIdx.x;
    corr_buf[corr_index] = hipCaddf(corr_buf[corr_index],c[1]);
    
    // don't store redundant pairs
    if (++j > blockIdx.x) return;

    corr_index = INTMULT(j,ninp) - INTMULT(j,(j+1))/2 + blockIdx.x;
    corr_index = INTMULT(corr_index,nchan) + threadIdx.x;
    corr_buf[corr_index] = hipCaddf(corr_buf[corr_index],c[2]);
    
    // don't store redundant pairs
    if (++j > blockIdx.x) return;

    corr_index = INTMULT(j,ninp) - INTMULT(j,(j+1))/2 + blockIdx.x;
    corr_index = INTMULT(corr_index,nchan) + threadIdx.x;
    corr_buf[corr_index] = hipCaddf(corr_buf[corr_index],c[3]);
}



/*  The GxG design uses shared memory to reduce the total number of global reads. The execution grid
    is (ninp/GRPSIZ) x (ninp/GRPSIZ) where GRPSIZ=2 or 4. The i index is the input index on the horizontal axis,
    the j index is the input index on the vertical axis and we want to compute the lower half of
    the correlation matrix. This means only compute products with i >= j.
    
    The kernel shares a value on the i axis between GRPSIZ products on the j axis. This means that
    instead of 2*GRPSIZ*GRPSIZ reads, we do 2*GRPSIZ read for GRPSIZ*GRPSIZ products.
    
    Each thread computes GRPSIZ correlation products. A thread reads a value on the i index and
    puts in shared memory to be used by itself and (GRPSIZ-1) other threads. It then reads

*/
/* this version hard coded for GRPSIZ =2 */
__global__ void do_CMAC_gpu_GxG(const int nchan, const int ninp, const int batchsize,
            hipFloatComplex * const ft_buf, hipFloatComplex * const corr_buf) {

    uint2 inp_ind;

    /* fetch the pre-calculated input indexes for this correlation product.
       this comes from constant cache, so is fast. */
    inp_ind = inp_index_gpu[blockIdx.y];

    /* shared i index data values */
    __shared__ hipFloatComplex d_i[2][MAX_CHAN/2];

    /* accumulators */
    hipFloatComplex c[2];
    c[0] = c[1] = make_hipFloatComplex(0.,0.);

    /* index of channel for this thread */
    const int chan = threadIdx.x + blockIdx.x*(MAX_CHAN/2);
    /* the start of the i,j indexes of interest are at inp_ind.x*GRPSIZ and inp_ind.y*GRPSIZ */
    const int j = inp_ind.y*2 + threadIdx.y; /* j index for thread */
    int       i = inp_ind.x*2 + threadIdx.y;

    /* loop over batches (successive time instants) */
    for (int batch=0; batch < batchsize; batch++) {
        /* data along j axis */
        hipFloatComplex d_j;

        /* fetch the input j value for this time instant */
        d_j = ft_buf[INTMULT(nchan,(INTMULT(ninp,batch) + j)) + chan];

        /* fetch the input i values for this time instant and conjugate */
        d_i[threadIdx.y][threadIdx.x] = hipConjf(ft_buf[INTMULT(nchan,(INTMULT(ninp,batch) + i)) + chan]);
        __syncthreads();

        /* do the CMAC */
        c[0] = hipCaddf(c[0],hipCmulf(d_i[0][threadIdx.x],d_j));
        c[1] = hipCaddf(c[1],hipCmulf(d_i[1][threadIdx.x],d_j));
        
        // another sync is necessary here so that all threads proceed to the next time batch together.
        __syncthreads();

#ifdef __DEVICE_EMULATION__
        if (batch==0) {
            printf("blk: %d,%d, thr: %d,%d, inps: %d,%d, i,j: %d,%d, chan: %d, d_j: (%g,%g), d_i: (%g,%g).",
                blockIdx.x,blockIdx.y,threadIdx.x,threadIdx.y, inp_ind.x, inp_ind.y, i,j,chan,
                hipCrealf(d_j),hipCimagf(d_j),
                hipCrealf(d_i[threadIdx.x][threadIdx.y]),hipCimagf(d_i[threadIdx.x][threadIdx.y]));
            printf(" c0: (%g,%g), c1: (%g,%g), c2: (%g,%g), c3: (%g,%g)\n",
                hipCrealf(c[0]),hipCimagf(c[0]),hipCrealf(c[1]),hipCimagf(c[1]),
                hipCrealf(c[2]),hipCimagf(c[2]),hipCrealf(c[3]),hipCimagf(c[3]));
        }
#endif
    }

    /* store accumulation results. Don't store redundant products. */
    /* the following formula decodes the correlation index (including autocorrelations) from the input
       indices i and j, for the conceptual bottom half of correlation matrix (i >= j products)
       corr_index = j*ninp - j*(j+1)/2 + i
       be careful with the divide by 2 not to cause rounding - don't factorize out j.
    */
    /* each thread has calculated 4 correlation products along the i axis, so we accumulate where
       i >= j */
    int corr_index;
    i = inp_ind.x*2; /* reset i back to start of 4x4 block */
    corr_index = j*ninp - j*(j+1)/2 + i;    /* calc the correlation index for lower triangle */
    corr_index = corr_index*nchan + chan;   /* turn this into array offset for the channel */

    for (i=0; i < 2; i++) {
        if (i+inp_ind.x*2 >= j) {
            corr_buf[corr_index] = hipCaddf(corr_buf[corr_index],c[i]);
#ifdef __DEVICE_EMULATION__
            printf("storing products. blk: %d,%d, thr: %d,%d, i,j,chan: %d,%d,%d, cind: %d, arr_ind: %d, val: %g,%g\n",
                blockIdx.x,blockIdx.y,threadIdx.x,threadIdx.y,i+inp_ind.x*2,j,chan,(corr_index-chan)/nchan,corr_index,
                (double)hipCrealf(corr_buf[corr_index]),(double)hipCimagf(corr_buf[corr_index]));
#endif
        }
        corr_index += nchan;
    }
#ifdef __DEVICE_EMULATION__
    fflush(stdout);
#endif
}


/* this version hard coded for G =4 */
__global__ void do_CMAC_gpu_GxG_4(const int nchan, const int ninp, const int batchsize,
            hipFloatComplex * const ft_buf, hipFloatComplex * const corr_buf) {

    uint2 inp_ind;

    /* fetch the pre-calculated input indexes for this correlation product.
       this comes from constant cache, so is fast. */
    inp_ind = inp_index_gpu[blockIdx.y];

    /* shared i index data values */
    __shared__ hipFloatComplex d_i[4][MAX_CHAN/4];

    /* accumulators */
    hipFloatComplex c[4];
    c[0] = c[1] = c[2] = c[3] = make_hipFloatComplex(0.,0.);

    /* index of channel for this thread */
    const int chan = threadIdx.x + blockIdx.x*(MAX_CHAN/4);
    /* the start of the i,j indexes of interest are at inp_ind.x*GRPSIZ and inp_ind.y*GRPSIZ */
    const int j = inp_ind.y*4 + threadIdx.y; /* j index for thread */
    int       i = inp_ind.x*4 + threadIdx.y;

    /* loop over batches (successive time instants) */
    for (int batch=0; batch < batchsize; batch++) {
        /* data along j axis */
        hipFloatComplex d_j;

        /* fetch the input i values for this time instant and conjugate */
        d_i[threadIdx.y][threadIdx.x] = hipConjf(ft_buf[INTMULT(nchan,(INTMULT(ninp,batch) + i)) + chan]);
        __syncthreads();

        /* putting this fetch second uses less registers for some reason */
        /* fetch the input j value for this time instant */
        d_j = ft_buf[INTMULT(nchan,(INTMULT(ninp,batch) + j)) + chan];

        /* do the CMAC */
        c[0] = hipCaddf(c[0],hipCmulf(d_i[0][threadIdx.x],d_j));
        c[1] = hipCaddf(c[1],hipCmulf(d_i[1][threadIdx.x],d_j));
        c[2] = hipCaddf(c[2],hipCmulf(d_i[2][threadIdx.x],d_j));
        c[3] = hipCaddf(c[3],hipCmulf(d_i[3][threadIdx.x],d_j));
        
        // another sync is necessary here so that all threads proceed to the next time batch together.
        __syncthreads();

#ifdef __DEVICE_EMULATION__
        if (batch==0) {
            printf("blk: %d,%d, thr: %d,%d, inps: %d,%d, i,j: %d,%d, chan: %d, d_j: (%g,%g), d_i: (%g,%g).",
                blockIdx.x,blockIdx.y,threadIdx.x,threadIdx.y, inp_ind.x, inp_ind.y, i,j,chan,
                hipCrealf(d_j),hipCimagf(d_j),
                hipCrealf(d_i[threadIdx.x][threadIdx.y]),hipCimagf(d_i[threadIdx.x][threadIdx.y]));
            printf(" c0: (%g,%g), c1: (%g,%g), c2: (%g,%g), c3: (%g,%g)\n",
                hipCrealf(c[0]),hipCimagf(c[0]),hipCrealf(c[1]),hipCimagf(c[1]),
                hipCrealf(c[2]),hipCimagf(c[2]),hipCrealf(c[3]),hipCimagf(c[3]));
        }
#endif
    }

    /* store accumulation results. Don't store redundant products. */
    /* the following formula decodes the correlation index (including autocorrelations) from the input
       indices i and j, for the conceptual bottom half of correlation matrix (i >= j products)
       corr_index = j*ninp - j*(j+1)/2 + i
       be careful with the divide by 2 not to cause rounding - don't factorize out j.
    */
    /* each thread has calculated 4 correlation products along the i axis, so we accumulate where
       i >= j */
    int corr_index;
    i = inp_ind.x*4; /* reset i back to start of 4x4 block */
    corr_index = j*ninp - j*(j+1)/2 + i;    /* calc the correlation index for lower triangle */
    corr_index = corr_index*nchan + chan;   /* turn this into array offset for the channel */

    for (i=0; i < 4; i++) {
        if (i+inp_ind.x*4 >= j) {
            corr_buf[corr_index] = hipCaddf(corr_buf[corr_index],c[i]);
#ifdef __DEVICE_EMULATION__
            printf("storing products. blk: %d,%d, thr: %d,%d, i,j,chan: %d,%d,%d, cind: %d, arr_ind: %d, val: %g,%g\n",
                blockIdx.x,blockIdx.y,threadIdx.x,threadIdx.y,i+inp_ind.x*4,j,chan,(corr_index-chan)/nchan,corr_index,
                (double)hipCrealf(corr_buf[corr_index]),(double)hipCimagf(corr_buf[corr_index]));
#endif
        }
        corr_index += nchan;
    }
#ifdef __DEVICE_EMULATION__
    fflush(stdout);
#endif
}


/*  1x1 pair parallel design is the simplest CMAC method.
    ninp*(ninp+1)/2 thread blocks are started. The thread decodes its i and j index
    from the pre-calculated lookup table in inp_index_gpu. Each thread fetches
    the value for the i and j index and calculates the product, summing over several
    time intervals. The results is added into the accumulator in global memory at
    the end.
*/
__global__ void do_CMAC_gpu_1x1(const int nchan, const int ninp, const int batchsize,
            hipFloatComplex * const ft_buf, hipFloatComplex * const corr_buf) {

    /* inputs (a,b) and result (c) */
    hipFloatComplex c;
    uint2 inp_ind;

    /* fetch the pre-calculated input indexes for this correlation product.
       this comes from constant cache, so is fast. */
    inp_ind = inp_index_gpu[blockIdx.x];

    c = make_hipFloatComplex(0.0,0.0);

    /* loop over all the time indices (one for each FFT batch) since they all go into the same product */
    for(int batch_ind=0; batch_ind<batchsize; batch_ind++) {

        hipFloatComplex a,b;

        a = ft_buf[INTMULT(INTMULT(batch_ind,ninp)+inp_ind.x,nchan) + threadIdx.x];

        if (inp_ind.x==inp_ind.y) {
            // don't re-fetch for autocorrelations. It is slightly faster this way.
            b = hipConjf(a);
        } else {
            b = hipConjf(ft_buf[INTMULT(INTMULT(batch_ind,ninp)+inp_ind.y,nchan) + threadIdx.x]);
        }

#ifdef __DEVICE_EMULATION__
        //if (batch_ind==0) printf("blk: %d, thr: %d, cind: %d, batch: %d, inp1: %d, inp2: %d\n",
        //   blockIdx.x,threadIdx.x,blockIdx.x,batch_ind,(int)inp_ind.x,(int)inp_ind.y);
#endif
        /* do the complex mult (conjugation already done above) */
        c = hipCaddf(c,hipCmulf(a,b));

    }

    /* accumulate and store result */
    int corr_index = INTMULT(blockIdx.x,nchan) + threadIdx.x;
    corr_buf[corr_index] = hipCaddf(corr_buf[corr_index],c);
}


/* initialise the input index lookup table for 1x1 CMAC */
int init_CMAC_1x1(void) {
    int i,j,cindex=0;
    uint2 inp_index_cpu[MAX_CORR_PROD];
    hipError_t res;

    /* zero entire array first */
    memset(inp_index_cpu, '\0', sizeof(inp_index_cpu));

    for(i=0; i<ninp; i++) {
        for (j=i; j<ninp; j++){
            inp_index_cpu[cindex] = make_uint2(i,j);
            cindex++;
        }
    }
    res = hipMemcpyToSymbol(HIP_SYMBOL(inp_index_gpu),inp_index_cpu,sizeof(inp_index_cpu));
    if (res != hipSuccess) {
        fprintf(stderr,"Error on memcpy of inp_index to device. Message: %s\n",hipGetErrorString(res));
        return res;
    }
    return 0;
}


/* initialise the input index lookup table for GxG CMAC */
int init_CMAC_GxG(const int grpsize) {
    int i,j,cindex=0;
    uint2 inp_index_cpu[MAX_CORR_PROD];
    hipError_t res;
    
    /* zero entire array first */
    memset(inp_index_cpu, '\0', sizeof(inp_index_cpu));

    for(j=0; j<ninp/grpsize; j++) {
        for (i=j; i<ninp/grpsize; i++){
            inp_index_cpu[cindex] = make_uint2(i,j);
            cindex++;
        }
    }

    res = hipMemcpyToSymbol(HIP_SYMBOL(inp_index_gpu),inp_index_cpu,sizeof(inp_index_cpu));
    if (res != hipSuccess) {
        fprintf(stderr,"Error on memcpy of inp_index to device. Message: %s\n",hipGetErrorString(res));
        return res;
    }
    return 0;
}


/* execute the FFT on the GPU. The input and output buffer pointers must be for memory
   on the GPU device, not the host machine. FFTs are batched here together to avoid
   overhead of calling GPU and to maximise GPU utilisation. */
int do_FFT_gpu(int nchan, int ninp, hipfftComplex *inp_buf, hipfftComplex *ft_buf) {
    static hipfftHandle plan=0;
    static int doneplan=0;

    hipfftResult res;
    
    /* make the FFTW execution plans. The CUDA FFT can do a batch of 1D FFTs at the same time. (sweet!)
       memory must be a contiguous block the size of siz_inp_buf*fft_batchsize for input
       and ninp*(nchan)*sizeof(complex) for result */
    if (!doneplan) {
        
        res = hipfftPlan1d(&plan,nchan,HIPFFT_C2C,ninp*fft_batchsize);
        if (res != HIPFFT_SUCCESS) {
            fprintf(stderr,"ERROR: hipfftPlan1d failed with error code %d\n",res);
            return res;
        }
        if (debug) fprintf(stderr,"Made a plan (id: %d) for %d channel FFT with batch size %d*%d\n",
                        (int) plan,nchan,ninp,fft_batchsize);
        doneplan=1;
    }
    
    res = hipfftExecC2C(plan,inp_buf,ft_buf,HIPFFT_FORWARD);
    if (res != HIPFFT_SUCCESS) {
        fprintf(stderr,"cuda FFT failed with result code %d\n",res);
        return res;
    }
    return 0;
}

/* write out correlation products.
   Apply a normalisation factor that depends on the FFT length and the number
   of averages so the flux density is the same regardless of the spectral channel width
   NOTE: Complex input voltages contain negative and positive frequencies, so a spectrum
   of bandwidth B goes from -B/2 to B/2 freq. So we need to shift the output channel indices
   when writing out. In the FFT, channel 0 is the center of the band. Channel N/2 is the
   start of the band, going up to N, then wrapping around back to channel 0, up to N/2-1.
*/
void writeOutput(FILE *fout_ac, FILE *fout_cc,int ninp, int nchan, int naver, int prod_type,
                hipfftComplex *buf,float normaliser) {
    int inp1,inp2,cprod=0,chan,index;
    float *temp_buffer=NULL;

    temp_buffer = (float *)malloc(sizeof(float)*(nchan));

    for(inp1=0; inp1<ninp; inp1++) {
        for (inp2=inp1; inp2<ninp; inp2++) {
        	index = cprod*nchan;
            /* make an average by dividing by the number of chunks that went into the total */
            for (chan=0; chan<nchan; chan++) {
                buf[index+chan] = make_hipFloatComplex(hipCrealf(buf[index+chan])*normaliser,hipCimagf(buf[index+chan])*normaliser);
                /* convert the autocorrelation numbers into floats, since the imag parts will be zero*/
                if (inp1==inp2 && (prod_type == 'A' || prod_type=='B')){
                    temp_buffer[chan] = hipCrealf(buf[index+chan]);
                }
            }
            if(inp1==inp2 && (prod_type == 'A' || prod_type=='B')) {
                /* write the auto correlation product */
                fwrite(temp_buffer+nchan/2,sizeof(float),nchan/2,fout_ac);
                fwrite(temp_buffer,sizeof(float),nchan/2,fout_ac);
            }
            if(inp1!=inp2 && (prod_type == 'C' || prod_type=='B')) {
                /* write the cross correlation product */
                fwrite(buf+index+nchan/2,sizeof(hipfftComplex),nchan/2,fout_cc);
                fwrite(buf+index,sizeof(hipfftComplex),nchan/2,fout_cc);
            }

            /* reset the correlation products to zero */
            memset(buf+index,'\0',(nchan)*sizeof(hipfftComplex));
            cprod++;
        }
    }
    if (temp_buffer!=NULL) free(temp_buffer);
}


/* incoming data is a stream of samples, one per input per time sample. This needs to be packed into arrays
   where nchan values for the same input channel are contiguous. Also, for FFT batching, we load fft_batchsize
   time chunks (1 chunk = nchan values per channel) at a time. This then means we have fft_batchsize sets of
   ninp sets of nchan samples per call to this function */
int readData(int nchan,int ninp,FILE *fpin,unsigned char *inp_buf) {
    int ntoread=0,nread;

    ntoread = ninp*nchan*fft_batchsize;
    
    nread = fread(inp_buf,wordsize,ninp*nchan*fft_batchsize,fpin);
    if(nread < ntoread) return 1;
            
    return 0;
}


/* open the input and output files */
int openFiles(char *infilename, char *outfilename, int prod_type, FILE **fin, FILE **fout_ac, FILE **fout_cc) {
    char tempfilename[FILENAME_MAX];
    int res=0;
    //char *inputbuf=NULL;
    
    if (infilename == NULL) {
        fprintf(stderr,"No input file specified\n");
        exit(1);
    }
    if (outfilename == NULL) {
        fprintf(stderr,"No output file specified\n");
        exit(1);
    }
    
    /* sanity check: can only use stdout for one type of output */
    if((prod_type=='B') && strcmp(outfilename,"-")==0) {
        fprintf(stderr,"Can only use stdout for either auto or cross correlations, not both\n");
        exit(1);
    }
    
    /* check for special file name: "-", which indicates to use stdin/stdout */
    if (strcmp(infilename,"-")==0) {
        *fin = stdin;
    } else {
        *fin = fopen(infilename,"r");
        if (*fin ==NULL) {
            fprintf(stderr,"failed to open input file name: <%s>\n",infilename);
            exit(1);
        }
    }
    /* if input is a pipe, see if we can increase its size */
    res=fcntl(fileno(*fin),F_GETPIPE_SZ,0);
    if (res > 0) {
        if (debug) {
            fprintf(stderr,"Current input pipe size: %d\n",res);
        }
        res=fcntl(fileno(*fin),F_SETPIPE_SZ,1048576);
        res=fcntl(fileno(*fin),F_GETPIPE_SZ,0);
         if (debug) {
            fprintf(stderr,"New input pipe size: %d\n",res);
        }
    }
    
    if ((prod_type=='A') && strcmp(outfilename,"-")==0) {
        *fout_ac = stdout;
    } else if ((prod_type=='C') && strcmp(outfilename,"-")==0) {
        *fout_cc = stdout;
    } else {
        if (prod_type=='A' || prod_type=='B') {
            strncpy(tempfilename,outfilename,FILENAME_MAX-8);
            strcat(tempfilename,".LACSPC");
            *fout_ac = fopen(tempfilename,"w");
            if (*fout_ac ==NULL) {
                fprintf(stderr,"failed to open output file name: <%s>\n",tempfilename);
                exit(1);
            }
        } 
        if (prod_type=='C' || prod_type=='B') {
            strncpy(tempfilename,outfilename,FILENAME_MAX-8);
            strcat(tempfilename,".LCCSPC");
            *fout_cc = fopen(tempfilename,"w");
            if (*fout_cc ==NULL) {
                fprintf(stderr,"failed to open output file name: <%s>\n",tempfilename);
                exit(1);
            }
        } 
    }
    
    return 0;
}


void parse_cmdline(int argc, char * const argv[], const char *optstring) {
    int c;
    
    while ((c=getopt(argc,argv,optstring)) != -1) {
        switch(c) {
            case 'c':
                nchan = atoi(optarg);
                if (nchan <=0 || nchan > MAX_CHAN || nchan %8 !=0) {
                    fprintf(stderr,"bad number of channels: %d. Max: %d. Must be power of 2\n",nchan,MAX_CHAN);
                    print_usage(argv);
                }
                break;
            case 'n':
                ninp = atoi(optarg);
                if (ninp <=0 || ninp > MAX_INPUTS) {
                    fprintf(stderr,"bad number of inputs: %d\n",ninp);
                    print_usage(argv);
                }
                break;
            case 'a':
                naver = atoi(optarg);
                if (naver <=0 || naver > 1000000) {
                    fprintf(stderr,"bad number of averages: %d\n",naver);
                    print_usage(argv);
                }
                break;
            case 'm':
                cmac_method = atoi(optarg);
                if (cmac_method < 0 || cmac_method >= CMAC_END) {
                    fprintf(stderr,"Unknown CMAC method: %d\n",cmac_method);
                    print_usage(argv);
                }
                break;
            case 'f':
                fft_batchsize = atoi(optarg);
                if (fft_batchsize <=0 || fft_batchsize > 1024) {
                    fprintf(stderr,"bad fft_batchsize: %d\n",naver);
                    print_usage(argv);
                }
                break;
            case 'i':
                infilename=optarg;
                break;
            case 'o':
                outfilename=optarg;
                break;
            case 'w':
                wordtype=atoi(optarg);
                if (wordtype < 0 || wordtype > DATATYPE_COMPLEX_SIGNED_BYTE) {
                    fprintf(stderr,"Bad data word type: %d\n",wordtype);
                    print_usage(argv);
                }
                if (wordtype==DATATYPE_MWA16)          wordsize=2;
                if (wordtype==DATATYPE_MWA8)           wordsize=1;
                if (wordtype==DATATYPE_COMPLEX_SHORT)  wordsize=4;
                if (wordtype==DATATYPE_COMPLEX_FLOAT)  wordsize=8;
                if (wordtype==DATATYPE_COMPLEX_UNSIGNED_BYTE || 
                    wordtype==DATATYPE_COMPLEX_SIGNED_BYTE) wordsize=2;
                if (wordtype==DATATYPE_UNSIGNED_BYTE || wordtype==DATATYPE_SIGNED_BYTE) {
                    wordsize=1;
                    complex_data=0;
                }
                if (wordtype==DATATYPE_UNSIGNED_SHORT|| wordtype==DATATYPE_SIGNED_SHORT) {
                    wordsize=2;
                    complex_data=0;
                }
                if (wordtype==DATATYPE_FLOAT) {
                    wordsize=4;
                    complex_data=0;
                }
                if (wordtype==DATATYPE_REAL_UNSIGNED_4BIT) {
                    fprintf(stderr,"Data word type: %d not supported\n",wordtype);
                    complex_data=0;
                    exit(1);
                }

                break;
            case 'u':
                unpack_method=atoi(optarg);
                if (unpack_method < 0 || unpack_method > 1) {
                    fprintf(stderr,"Bad unpack method: %d\n",unpack_method);
                    print_usage(argv);
                }
                break;
            case 'p':
                prod_type = toupper(optarg[0]);
                if (prod_type!='A' && prod_type !='B' && prod_type != 'C') {
                    fprintf(stderr,"bad correlation product type: %c\n",prod_type);
                    print_usage(argv);
                }
                break;
            default:
                fprintf(stderr,"unknown option %c\n",c);
                print_usage(argv);
        }
    }
    if (wordsize==0) {
        fprintf(stderr,"wordsize is 0\n");
        exit(1);
    }
}

/* returns the elapsed wall-clock time, in ms, since start (without resetting start) */
static float elapsed_time(struct timeval *start){
    struct timeval now;
    gettimeofday(&now,NULL);
    return 1.e3f*(float)(now.tv_sec-start->tv_sec) +
        1.e-3f*(float)(now.tv_usec-start->tv_usec);
}


void printGPUDetails(FILE *fp) {
    hipError_t res;
    int numdev=0;
    struct hipDeviceProp_t devprop;
    
    res = hipGetDeviceCount(&numdev);
    if (res != 0) {
        fprintf(fp,"failed to get number of CUDA devices\n");
        exit(1);
    }
    fprintf(fp,"There are %d devices.\n",numdev);
    
    for(int dev=0; dev< numdev; dev++) {
        res = hipGetDeviceProperties(&devprop,dev);
        if (res != 0) {
            fprintf(fp,"failed to get properties for device %d\n",dev);
            exit(1);
        }
    
        fprintf(fp,"Device %d:\nname:\t\t%s\n",dev,devprop.name);
        fprintf(fp,"MEM:\t\t%ld MB\n",(long)(devprop.totalGlobalMem)/(1024*1024));
        fprintf(fp,"Shmem/block:\t%ld\n",(long)devprop.sharedMemPerBlock);
        fprintf(fp,"regs/block:\t%ld\n",(long)devprop.regsPerBlock);
        fprintf(fp,"Warp size:\t%d\n",devprop.warpSize);
        fprintf(fp,"Mem pitch:\t%ld\n",(long)devprop.memPitch);
        fprintf(fp,"Max thr/blk:\t%d\n",devprop.maxThreadsPerBlock);
        fprintf(fp,"Max dim/blk:\t%d,%d,%d\n",devprop.maxThreadsDim[0],devprop.maxThreadsDim[1],devprop.maxThreadsDim[2]);
        fprintf(fp,"Tot const mem:\t%ld\n",(long)devprop.totalConstMem);
        fprintf(fp,"Version:\t%d.%d\n",devprop.major,devprop.minor);
        fprintf(fp,"clockrate:\t%d\n",devprop.clockRate);
        fprintf(fp,"texture algn:\t%ld\n",(long)devprop.textureAlignment);
    }
}


void print_usage(char * const argv[]) {
    fprintf(stderr,"Usage:\n%s [options]\n",argv[0]);
    fprintf(stderr,"\t-p type\t\tspecify correlation product type(s). A: auto, C: cross, B: both. default: %c\n",prod_type);
    fprintf(stderr,"\t-c num\t\tspecify number of freq channels. default: %d\n",nchan);
    fprintf(stderr,"\t-n num\t\tspecify number of input streams. default: %d\n",ninp);
    fprintf(stderr,"\t-a num\t\tspecify number of averages before output. default: %d\n",naver);
    fprintf(stderr,"\t-f num\t\tspecify fft batchsize. default: %d\n",fft_batchsize);
    fprintf(stderr,"\t-w num\t\tspecify data word type. Default: %d\n",wordtype);
    fprintf(stderr,"\t      \t%d: MWA 5+5bit real/imag encoded in 16 bits.\n",DATATYPE_MWA16);
    fprintf(stderr,"\t      \t%d: MWA 4+4bit real/imag encoded in 8 bits.\n",DATATYPE_MWA8);
    fprintf(stderr,"\t      \t%d: Native 16+16 bit signed int real/imag.\n",DATATYPE_COMPLEX_SHORT);
    fprintf(stderr,"\t      \t%d: Native 32+32 bit float complex real/imag.\n",DATATYPE_COMPLEX_FLOAT);
    fprintf(stderr,"\t      \t%d: Signed byte (real only).\n",DATATYPE_SIGNED_BYTE);
    fprintf(stderr,"\t      \t%d: Unsigned byte (real only).\n",DATATYPE_UNSIGNED_BYTE);
    fprintf(stderr,"\t      \t%d: Signed short (real only),\n",DATATYPE_SIGNED_SHORT);
    fprintf(stderr,"\t      \t%d: Unsigned short (real only),\n",DATATYPE_UNSIGNED_SHORT);
    fprintf(stderr,"\t      \t%d: Single precision float (real only),\n",DATATYPE_FLOAT);
    fprintf(stderr,"\t      \t%d: Native 8+8 bit unsigned int (complex),\n",DATATYPE_COMPLEX_UNSIGNED_BYTE);
    fprintf(stderr,"\t      \t%d: Native 8+8 bit signed int (complex),\n",DATATYPE_COMPLEX_SIGNED_BYTE);
    fprintf(stderr,"\t-u num\t\tspecify data unpack & corner turn method. (0=standard, 1=transpose) Default: %d\b\n",unpack_method);
    fprintf(stderr,"\t-i filename\tinput file name. use '-' for stdin\n");
    fprintf(stderr,"\t-o filename\toutput file name. use '-' for stdout\n");
    fprintf(stderr,"\t-m num\t\tspecify CMAC method. 0: 1x1, 1: 1xG=2, 2: 1xG=4, 3: GxG=2, 4: GxG=4. Default: %d\n",cmac_method);
    exit(0);
}

